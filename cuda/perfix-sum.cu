#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__ void fill_one(int* d_array, size_t length) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index >= length ) {
        return;
    }
    d_array[index] = 1;
}

#define BLOCK_SIZE 2 

__global__ void perfix_sum_simple(int* d_array, size_t length) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("index %d length %d", index, length);
    if ( index >= length ) {
        return;
    }
    __shared__ int cache[BLOCK_SIZE];

    cache[threadIdx.x] = d_array[index];

    for ( size_t stride = 1; stride <= threadIdx.x; stride *= 2 ) {
        __syncthreads();
        cache[threadIdx.x] += cache[threadIdx.x - stride];
    }
    // write back
    d_array[index] = cache[threadIdx.x];
}

__global__ void perfix_sum( int* d_array, size_t block_size, size_t length) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    const int start = index * block_size;

    if ( start >= length ) {
        return;
    }
    __shared__ int cache[BLOCK_SIZE];
    int local_copy[BLOCK_SIZE];

    for ( size_t i = 0; i < block_size; ++i ) {
        local_copy[i] = d_array[ start + i ];
    }

    for ( size_t stride = 1; stride < BLOCK_SIZE; stride *= 2 ) {
        cache[threadIdx.x] = local_copy[block_size-1];
        __syncthreads();
        int operend = cache[threadIdx.x-stride];
        for ( size_t i = 0; i < block_size; ++i ) {
            local_copy[i] += operend;
        }
    }

    // write back
    for ( size_t i = 0; i < block_size; ++i ) {
        d_array[ start + i ] = local_copy[i];
    }
}

#define BLOCK_NUM 1

int main(int argc, char** argv) {
    int* d_array = NULL; 
    checkCudaErrors(hipMalloc(&d_array, sizeof(int) * BLOCK_SIZE * BLOCK_NUM));

    fill_one<<<BLOCK_NUM, BLOCK_SIZE>>>(d_array, BLOCK_SIZE * BLOCK_NUM);

    perfix_sum<<<BLOCK_NUM, BLOCK_SIZE>>>(d_array, 1, BLOCK_SIZE * BLOCK_NUM);
    hipDeviceSynchronize();

    int h_array[BLOCK_NUM*BLOCK_SIZE] = {0};

    checkCudaErrors(hipMemcpy(h_array, d_array, sizeof(int) * BLOCK_SIZE * BLOCK_NUM, hipMemcpyDeviceToHost));

    for ( size_t i = 0; i < BLOCK_NUM * BLOCK_SIZE; ++i ) {
        std::cout << h_array[i] << " ";
        if ( (i % BLOCK_SIZE) == (BLOCK_SIZE-1) ) {
            std::cout << std::endl;
        }
    }

    checkCudaErrors(hipFree(d_array));
}
